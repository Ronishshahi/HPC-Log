#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *attempt) {
  char password1[] = "FT81";
  char password2[] = "CV07";
  char password3[] = "IR03";
  char password4[] = "SA31";

  char *r = attempt;
  char *o = attempt;
  char *n = attempt;
  char *h = attempt;
  char *pwd1 = password1;
  char *pwd2 = password2;
  char *pwd3 = password3;
  char *pwd4 = password4;

  while(*r == *pwd1) {
   if(*r == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    r++;
    pwd1++;
  }
    
  while(*o == *pwd2) {
   if(*o == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    o++;
    pwd2++;
  }

  while(*n == *pwd3) {
   if(*n == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    n++;
    pwd3++;
  }

  while(*h == *pwd4) {
   if(*h == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    h++;
    pwd4++;
  }
  return 0;

}

__global__ void  kernel() {
char s,z;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(s='0'; s<='9'; s++){
      for(z='0'; z<='9'; z++){
            password[2] = s;
            password[3] = z;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}




